#include "hip/hip_runtime.h"
//
// Author: Eduardo Aponte
// Copyright 2015 by Eduardo Aponte <aponteeduardo@gmail.com>
//
// Licensed under GNU General Public License 3.0 or later.
// Some rights reserved. See COPYING, AUTHORS.
//
// Revision log:
//


#include "mpdcm.hcu"

// General functions

__device__
MPFLOAT
dcm_dx(dbuff x, dbuff y, dbuff u, void *p_theta,
     void *p_ptheta, int i)
{
    MPFLOAT dx = 0;
    MPFLOAT bt = 0;
    int nx = x.dim;
    int j;
    int k;
    int l;
    int o;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    o = INDEX_X * x.dim;

    // A

    for (j = 0; j < nx; j++)
    {
        bt = 0;
        if ( theta->fD == MF_TRUE )
        {
            for ( l = 0; l < nx; l++ )
            {
                int o = (nx + 1) * j;
                int oj = theta->sB->j[o + l]; 
                for (k = 0; k < theta->sD->j[o + l + 1] - oj;  k++)
                {
                    if ( theta->sD->i[oj + k] == i )
                    {
                        bt = fma(x.arr[l], theta->sD->v[oj + k], bt);
                    }
                }
            }
           
            bt = 0;
            k = nx * nx * j + i;
            for (l = 0; l < x.dim; l++)
                bt = fma(theta->D[k + nx * l], x.arr[o + l], bt); 
        }
        dx = fma(theta->A[i + nx*j] + bt, x.arr[o + j], dx);
    }

    for (j = 0; j < u.dim; j++)
    {
        if (  u.arr[j] == 0  )
            continue;

        bt = 0;
        for (l = 0; l < nx; l ++)
        {
            int o = (nx + 1) * j;
            int oj = theta->sB->j[o + l]; 
            for (k = 0; k < theta->sB->j[o + l + 1] - oj;  k++)
            {
                if ( theta->sB->i[oj + k] == i )
                {
                    bt = fma(x.arr[l], theta->sB->v[oj + k], bt);
                }
            }
        }

        // C

        dx = fma(theta->C[i + x.dim * j] + bt, u.arr[j], dx);
    }
    return dx;
}

__device__ 
MPFLOAT 
dcm_ds(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    MPFLOAT ds;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    ds = x.arr[INDEX_X * x.dim + i] - 
        theta->K[i] * x.arr[INDEX_S * x.dim + i] -
        theta->gamma * (exp(x.arr[INDEX_F * x.dim + i]) - 1);

    return ds;
}

__device__
MPFLOAT
dcm_df(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    MPFLOAT df;

    //ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    df = x.arr[INDEX_S * x.dim + i]*exp(-x.arr[INDEX_F * x.dim + i]);

    return df;
}

__device__
MPFLOAT 
dcm_dv(dbuff x, dbuff y, dbuff u, void *p_theta,
    void *p_ptheta, int i)
{
    MPFLOAT dv;

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dv = exp(x.arr[INDEX_F * x.dim + i] - x.arr[INDEX_V * x.dim + i] - 
            theta->tau[i]) -
        exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);

    return dv;
}

__device__
MPFLOAT
dcm_dq(dbuff x, dbuff y, dbuff u, void *p_theta, 
    void *p_ptheta, int i)
{
    MPFLOAT dq = 0;
    MPFLOAT f = exp(-x.arr[INDEX_F * x.dim + i]);
    MPFLOAT v;
    MPFLOAT lnE0; 
    ThetaDCM *theta = (ThetaDCM *) p_theta;
    
    v = exp(x.arr[INDEX_V * x.dim + i] * theta->alpha - theta->tau[i]);
    lnE0 = theta->lnE0;

    //    PThetaDCM *ptheta = (PThetaDCM  *) p_ptheta;

    dq = (1 - exp(f * theta->ln1_E0))*exp(x.arr[INDEX_F * x.dim + i] -
        lnE0 - theta->tau[i] -  x.arr[INDEX_Q * x.dim + i]) - v;

    return dq;
}

__device__ 
MPFLOAT
dcm_lk1(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;
    MPFLOAT q = exp(x.arr[INDEX_Q * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k1 * ( 1 - q);

    return l;
}


__device__
MPFLOAT
dcm_lk2(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;
    MPFLOAT qv = exp(x.arr[INDEX_Q * x.dim + i] - x.arr[INDEX_V *x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k2 * ( 1 - qv);

    return l;
}

__device__
MPFLOAT
dcm_lk3(dbuff x, dbuff y, dbuff u, void *p_theta,
            void *p_ptheta, int i)
{
    MPFLOAT l;

    MPFLOAT v = exp(x.arr[INDEX_V * x.dim + i]);

    ThetaDCM *theta = (ThetaDCM *) p_theta;
    //PThetaDCM *ptheta = (PThetaDCM *) p_ptheta;

    l = theta->k3 * ( 1 - v);

    return l;
}

__device__
void
dcm_upy(dbuff ox, dbuff y, dbuff u, void *theta,
    void *ptheta, dbuff nx)
{
    // Network node
    int j = threadIdx.x%y.dim;

    switch( threadIdx.y )
    {
        case INDEX_LK1 :
            nx.arr[ INDEX_LK1 * nx.dim + j] =
                dcm_lk1(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK2:
            nx.arr[ INDEX_LK2 * nx.dim + j] =
                dcm_lk2(ox, y, u, theta, ptheta, j);
            break;
        case INDEX_LK3 :
            nx.arr[ INDEX_LK3 * nx.dim + j] =
                dcm_lk3(ox, y, u, theta, ptheta, j);
            break;
    }

}


